# include <stdio.h>
# include "hip/hip_runtime.h"


#define N 1000000000

void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

__global__ void gpu_vector_add(float *out, float *a, float *b, int n) {
    int i = threadIdx.x;
    out[i] = a[i] + b[i];
}

__global__ void hello_cuda() {
    printf("Hello CUDA world\n");
}

int main(){
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; 
        b[i] = 2.0f;
    }

    // c function
    vector_add(out, a, b, N);

    // cuda function
    // gpu_vector_add<<<1,1>>>(out, a, b, N);
    hello_cuda<<<2, 20>>>();
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}
